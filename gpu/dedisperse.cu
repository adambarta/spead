#include <stdio.h>
#include <math.h>

#include <hipfft/hipfft.h>

#include <spead_api.h>


int spead_api_callback(struct spead_item_group *ig)
{
 
#ifdef DEBUG
  fprintf(stderr, "we are in the callback\n");
#endif

  uint64_t off = 0;
  uint64_t count;

  struct spead_api_item *itm;

  while (off < ig->g_size){
  
    itm = (struct spead_api_item *) (ig->g_map + off);

    if (itm->i_len == 0)
      goto skip;

    count = 0;
#ifdef DEBUG
    fprintf(stderr, "ITEM id[%d] vaild [%d] len [%ld]\n", itm->i_id, itm->i_valid, itm->i_len);
#endif
    
    print_data(itm->i_data, itm->i_len);

skip:
    off += sizeof(struct spead_api_item) + itm->i_len;
    
  }



  return 0;
}


