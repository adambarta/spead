



#include <hipfft/hipfft.h>




int main(int argc, char *argv[])
{
  
#define NX    256
#define BATCH 10

  hipfftHandle    plan;
  hipfftComplex  *data;
  hipMalloc((void **) &data, sizeof(hipfftComplex) * NX * BATCH);
  
  if (hipGetLastError() != hipSuccess){
#ifdef DEBUG
    fprintf(stderr, "cuda err: failed to allocate\n");
#endif
    return 1;
  }

  if (hipfftPlan1d(&plan, NX, HIPFFT_C2C, BATCH) != HIPFFT_SUCCESS) {
#ifdef DEBUG
    fprintf(stderr, "cuda err: plan creation failed\n");
#endif
    return 1;
  }

  if (hipfftExecC2C(plan, data, data, HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
#ifdef DEBUG
    fprintf(stderr ,"CUFFT error: ExecC2C Forward failed");
#endif
    return 1;
  }

  if (hipfftExecC2C(plan, data, data, HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
#ifdef DEBUG
    fprintf(stderr ,"CUFFT error: ExecC2C Forward failed");
#endif
    return 1;
  }
  
  if (hipDeviceSynchronize() != hipSuccess){
#ifdef DEBUG
    fprintf(stderr ,"CUFFT error: ExecC2C Forward failed");
#endif
    return 1;
  }

  hipfftDestroy(plan);
  hipFree(data);

  return 0;
}
