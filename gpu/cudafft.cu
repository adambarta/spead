#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <hipfft/hipfft.h>
#define PI (3.141592653589793)

#if 0
__global__ void real2complex(float *f, hipfftComplex *fc, int N)
{
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  int j = threadIdx.y + blockIdx.y*blockDim.y;

  int index = j*N+i;

  if (i<N && j<N) {
    fc[index].x = f[index];
    fc[index].y = 0.0f;
  }
}

__global__ void complex2real(hipfftComplex *fc, float *f, int N)
{
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  int j = threadIdx.y + blockIdx.y*blockDim.y;

  int index = j*N+i;
  
  if (i<N && j<N) {
    f[index] = fc[index].x/((float)N*(float)N);
    //divide by number of elements to recover value
  }
}
#endif

int main(int argc, char *argv[])
{
#define NX    256
#define BATCH 100

  int i; 

  hipfftHandle    plan;
  hipfftComplex  *odata;
  hipfftComplex  *cxdata;
  hipfftComplex  *redata;
  hipfftComplex  *idata;

  redata = (hipfftComplex*)malloc(sizeof(hipfftComplex)*NX*BATCH); 
  if (redata == NULL)
    return 1;

  cxdata = (hipfftComplex*)malloc(sizeof(hipfftComplex)*NX*BATCH); 
  if (cxdata == NULL)
    return 1;

  for (i=0; i<NX*BATCH; i++){
    redata[i].x = sinf(2 * PI * i / NX);
  }

#if 0
  for (i=0; i<NX*BATCH; i++){
#ifdef DEBUG
    fprintf(stderr, "%0.2f ", redata[i]);
#endif
  }
#ifdef DEBUG
  fprintf(stderr, "\n");
#endif
#endif

  hipMalloc((void **) &idata, sizeof(hipfftComplex) * NX * BATCH);
  if (hipGetLastError() != hipSuccess){
#ifdef DEBUG
    fprintf(stderr, "cuda err: failed to allocate\n");
#endif
    free(redata);
    free(cxdata);
    return 1;
  }

  hipMalloc((void **) &odata, sizeof(hipfftComplex) * NX * BATCH);
  if (hipGetLastError() != hipSuccess){
#ifdef DEBUG
    fprintf(stderr, "cuda err: failed to allocate\n");
#endif
    hipFree(idata);
    free(redata);
    free(cxdata);
    return 1;
  }

  hipMemcpy(idata, redata, sizeof(hipfftComplex) * NX * BATCH, hipMemcpyHostToDevice);
  if (hipGetLastError() != hipSuccess){
#ifdef DEBUG
    fprintf(stderr, "cuda err: failed copy\n");
#endif
    hipFree(idata);
    hipFree(odata);
    free(redata);
    free(cxdata);
    return 1;
  }

  if (hipfftPlan1d(&plan, NX, HIPFFT_C2C, BATCH) != HIPFFT_SUCCESS) {
#ifdef DEBUG
    fprintf(stderr, "cuda err: plan creation failed\n");
#endif
    hipFree(idata);
    hipFree(odata);
    free(redata);
    free(cxdata);
    return 1;
  }
  
  hipfftResult res = hipfftExecC2C(plan, idata, odata, HIPFFT_FORWARD);
  if (res != HIPFFT_SUCCESS) {
#ifdef DEBUG
    fprintf(stderr ,"CUFFT error: ExecC2C Forward failed %d\n", res);
#endif
    hipfftDestroy(plan);
    hipFree(idata);
    hipFree(odata);
    free(redata);
    free(cxdata);
    return 1;
  }

  hipMemcpy(cxdata, odata, sizeof(hipfftComplex) * NX * BATCH, hipMemcpyDeviceToHost);

#if 0
  if (hipfftExecC2C(plan, data, data, HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
#ifdef DEBUG
    fprintf(stderr ,"CUFFT error: ExecC2C Forward failed");
#endif
    hipfftDestroy(plan);
    hipFree(idata);
    hipFree(odata);
    free(redata);
    free(cxdata);
    return 1;
  }
#endif
  
  if (hipDeviceSynchronize() != hipSuccess){
#ifdef DEBUG
    fprintf(stderr ,"CUFFT error: ExecC2C Forward failed");
#endif
    hipfftDestroy(plan);
    hipFree(idata);
    hipFree(odata);
    free(redata);
    free(cxdata);
    return 1;
  }

  
  for (i=0; i<NX*BATCH; i++){
#ifdef DEBUG
    fprintf(stderr, "%d %0.5f %0.5f %0.5f\n", i, redata[i].x, hipCabsf(cxdata[i]), atan2(cxdata[i].y, cxdata[i].x));
#endif
  }
#ifdef DEBUG
  fprintf(stderr, "\n");
#endif

  hipfftDestroy(plan);
  hipFree(idata);
  hipFree(odata);
  free(redata);
  free(cxdata);

  return 0;
}
