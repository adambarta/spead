#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>

#include <hipfft/hipfft.h>
#define PI (3.141592653589793)

#if 0
__global__ void real2complex(float *f, hipfftComplex *fc, int N)
{
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  int j = threadIdx.y + blockIdx.y*blockDim.y;

  int index = j*N+i;

  if (i<N && j<N) {
    fc[index].x = f[index];
    fc[index].y = 0.0f;
  }
}

__global__ void complex2real(hipfftComplex *fc, float *f, int N)
{
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  int j = threadIdx.y + blockIdx.y*blockDim.y;

  int index = j*N+i;
  
  if (i<N && j<N) {
    f[index] = fc[index].x/((float)N*(float)N);
    //divide by number of elements to recover value
  }
}
#endif

int main(int argc, char *argv[])
{
#define NX    256
#define BATCH 10

  int i; 

  hipfftHandle    plan;
  hipfftComplex  *odata;
  hipfftComplex  *cxdata;
  hipfftReal     *redata;
  hipfftReal     *idata;

  redata = (hipfftReal*)malloc(sizeof(hipfftReal)*NX*BATCH); 
  if (redata == NULL)
    return 1;

  cxdata = (hipfftComplex*)malloc(sizeof(hipfftComplex)*NX*BATCH); 
  if (cxdata == NULL)
    return 1;

  for (i=0; i<NX*BATCH; i++){
    redata[i] = cosf(2 * PI * i / NX * BATCH);
  }

#if 0
  for (i=0; i<NX*BATCH; i++){
#ifdef DEBUG
    fprintf(stderr, "%0.2f ", redata[i]);
#endif
  }
#ifdef DEBUG
  fprintf(stderr, "\n");
#endif
#endif

  hipMalloc((void **) &idata, sizeof(hipfftReal) * NX * BATCH);
  if (hipGetLastError() != hipSuccess){
#ifdef DEBUG
    fprintf(stderr, "cuda err: failed to allocate\n");
#endif
    free(redata);
    free(cxdata);
    return 1;
  }

  hipMalloc((void **) &odata, sizeof(hipfftComplex) * NX * BATCH);
  if (hipGetLastError() != hipSuccess){
#ifdef DEBUG
    fprintf(stderr, "cuda err: failed to allocate\n");
#endif
    hipFree(idata);
    free(redata);
    free(cxdata);
    return 1;
  }

  hipMemcpy(idata, redata, sizeof(hipfftReal) * NX * BATCH, hipMemcpyHostToDevice);
  if (hipGetLastError() != hipSuccess){
#ifdef DEBUG
    fprintf(stderr, "cuda err: failed copy\n");
#endif
    hipFree(idata);
    hipFree(odata);
    free(redata);
    free(cxdata);
    return 1;
  }

  if (hipfftPlan1d(&plan, NX, HIPFFT_R2C, BATCH) != HIPFFT_SUCCESS) {
#ifdef DEBUG
    fprintf(stderr, "cuda err: plan creation failed\n");
#endif
    hipFree(idata);
    hipFree(odata);
    free(redata);
    free(cxdata);
    return 1;
  }
  
  hipfftResult res = hipfftExecR2C(plan, idata, odata);
  if (res != HIPFFT_SUCCESS) {
#ifdef DEBUG
    fprintf(stderr ,"CUFFT error: ExecC2C Forward failed %d\n", res);
#endif
    hipfftDestroy(plan);
    hipFree(idata);
    hipFree(odata);
    free(redata);
    free(cxdata);
    return 1;
  }

  hipMemcpy(cxdata, odata, sizeof(hipfftReal) * NX * BATCH, hipMemcpyDeviceToHost);


#if 0
  if (hipfftExecC2C(plan, data, data, HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
#ifdef DEBUG
    fprintf(stderr ,"CUFFT error: ExecC2C Forward failed");
#endif
    hipfftDestroy(plan);
    hipFree(idata);
    hipFree(odata);
    free(redata);
    free(cxdata);
    return 1;
  }
#endif
  
  if (hipDeviceSynchronize() != hipSuccess){
#ifdef DEBUG
    fprintf(stderr ,"CUFFT error: ExecC2C Forward failed");
#endif
    hipfftDestroy(plan);
    hipFree(idata);
    hipFree(odata);
    free(redata);
    free(cxdata);
    return 1;
  }


  for (i=0; i<NX*BATCH; i++){
#ifdef DEBUG
    fprintf(stderr, "%d: %0.2f %0.2f\n", i, cxdata[i].x, cxdata[i].y);
#endif
  }
#ifdef DEBUG
  fprintf(stderr, "\n");
#endif


  hipfftDestroy(plan);
  hipFree(idata);
  hipFree(odata);
  free(redata);
  free(cxdata);

  return 0;
}
