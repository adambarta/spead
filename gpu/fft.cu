#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <math.h>
#include <fcntl.h>

#include <hipfft/hipfft.h>

#include <spead_api.h>

#define NX      2048
#define BATCH   1

#define SPEAD_DATA_ID       0x0


struct cufft_o {
  hipfftHandle     plan;
  hipfftComplex    *d_in;
  hipfftComplex    *d_out;
};


void spead_api_destroy(void *data)
{
  struct cufft_o *fo;

  fo = data;

  if (fo){
  
    hipfftDestroy(fo->plan);
    
    if (fo->d_in){
      free(fo->d_in);
    }
    
    hipFree(fo->d_out);
  
  }
}


void *spead_api_setup()
{
  struct cufft_o *fo;
  
  fo = malloc(sizeof(struct cufft_o));
  if (fo == NULL){
#ifdef DEBUG
    fprintf(stderr, "e: logic could not malloc api obj\n");
#endif
    return NULL;
  }
  
  fo->d_in  = NULL;
  fo->d_out = NULL;
  
  if (hipfftPlan1d(&(fo->plan), NX, HIPFFT_C2C, BATCH) != HIPFFT_SUCCESS) {
#ifdef DEBUG
    fprintf(stderr, "cuda err: plan creation failed\n");
#endif
    spead_api_destroy(fo);
    return NULL;
  }

  fo->d_in = (hipfftComplex*) malloc(sizeof(hipfftComplex) * NX * BATCH); 
  if (fo->d_in == NULL){
#ifdef DEBUG
    fprintf(stderr, "e: malloc failed\n");
#endif
    spead_api_destroy(fo);
    return NULL;
  }
  
  hipMalloc((void **) &(fo->d_out), sizeof(hipfftComplex) * NX * BATCH);
  if (hipGetLastError() != hipSuccess){
#ifdef DEBUG
    fprintf(stderr, "cuda err: failed to cudamalloc\n");
#endif
    spead_api_destroy(fo);
    return NULL;
  }


  return fo;
}

int cufft_callback(struct cufft_o *fo, struct spead_api_item *itm)
{
  uint64_t i;

  uint8_t *d;  

  hipfftComplex *in;
  hipfftComplex *out;

  hipfftResult res;
  
  if (fo == NULL || itm == NULL){
    return -1;
  }
 
  if (NX*BATCH != itm->i_len){
#ifdef DEBUG
    fprintf(stderr, "e: data len [%ld] doesn't match fft setup NX*BATCH [%ld]\n", itm->i_len, (long int) NX*BATCH);
#endif
    return -1;
  }

  in  = fo->d_in;
  out = fo->d_out;
  d   = itm->i_data;

  if (in == NULL || out == NULL || d == NULL){
#ifdef DEBUG
    fprintf(stderr, "e: data pointers are null\n");
#endif
    return -1;
  }

  /*prepare the data into hipComplex*/
  
  for (i=0; i<NX*BATCH; i++){
    in[i].x = (float) d[i];
    in[i].y = 0;
  }
  
  hipMemcpy(out, in, sizeof(hipfftComplex) * NX * BATCH, hipMemcpyHostToDevice);
  if (hipGetLastError() != hipSuccess){
#ifdef DEBUG
    fprintf(stderr, "cuda err: failed copy\n");
#endif
    return -1;
  }

  res = hipfftExecC2C(fo->plan, out, out, HIPFFT_FORWARD);
  if (res != HIPFFT_SUCCESS) {
#ifdef DEBUG
    fprintf(stderr ,"CUFFT error: ExecC2C Forward failed %d\n", res);
#endif
    return -1;  
  }

  hipMemcpy(in, out, sizeof(hipfftComplex) * NX * BATCH, hipMemcpyDeviceToHost);
  if (hipGetLastError() != hipSuccess){
#ifdef DEBUG
    fprintf(stderr, "cuda err: failed copy\n");
#endif
    return -1;
  }

#if 0
  print_data( (unsigned char *) in, sizeof(hipfftComplex)*NX*BATCH);
  
  for (i=0; i<NX*BATCH; i++){
    fprintf(stderr, "%f + j %f\n", in[i].x, in[i].y);
  }
#endif

  for (i=0; i<NX*BATCH; i++){
#ifdef DEBUG
    fprintf(stdout, "%ld %0.5f\n", i, hipCabsf(in[i]));
#endif
  }
#ifdef DEBUG
  fprintf(stdout, "e\n");
#endif
  for (i=0; i<NX*BATCH; i++){
#ifdef DEBUG
    fprintf(stdout, "%ld %0.5f\n", i, atan2(in[i].y, in[i].x));
#endif
  }
#ifdef DEBUG
  fprintf(stdout, "e\n");
#endif
  
  return 0;
}


int spead_api_callback(struct spead_item_group *ig, void *data)
{
  struct spead_api_item *itm;
  struct cufft_o *fo;
  uint64_t off;

  fo = data;

  if (fo == NULL || ig == NULL){
#ifdef DEBUG
    fprintf(stderr, "e: NULL params for <%s>\n", __func__);
#endif
    return -1;
  }
  
  off = 0;
  while (off < ig->g_size){
    itm = (struct spead_api_item *) (ig->g_map + off);

#ifdef DEBUG
    fprintf(stderr, "ITEM id[0x%x] vaild [%d] len [%ld]\n", itm->i_id, itm->i_valid, itm->i_len);
#endif
    if (itm->i_len == 0)
      goto skip;

    if (itm->i_id == SPEAD_DATA_ID){
      break;
    }
skip:
    off += sizeof(struct spead_api_item) + itm->i_len;
  }

  if (itm->i_id != SPEAD_DATA_ID){
#ifdef DEBUG
    fprintf(stderr, "%s: err dont have requested data id\n", __func__);
#endif
    return -1;
  }
  

  if (cufft_callback(fo, itm) < 0){
    return -1;
  }


  return 0;
}


#if 0
#define PI (3.141592653589793)

__global__ void real2complex(float *f, hipfftComplex *fc, int N)
{
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  int j = threadIdx.y + blockIdx.y*blockDim.y;

  int index = j*N+i;

  if (i<N && j<N) {
    fc[index].x = f[index];
    fc[index].y = 0.0f;
  }
}

__global__ void complex2real(hipfftComplex *fc, float *f, int N)
{
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  int j = threadIdx.y + blockIdx.y*blockDim.y;

  int index = j*N+i;
  
  if (i<N && j<N) {
    f[index] = fc[index].x/((float)N*(float)N);
    //divide by number of elements to recover value
  }
}
#endif

#if 0
int main(int argc, char *argv[])
{
#define NX    256
#define BATCH 100

  int i; 

  hipfftHandle    plan;
  hipfftComplex  *odata;
  hipfftComplex  *cxdata;
  hipfftComplex  *redata;
  hipfftComplex  *idata;

  redata = (hipfftComplex*)malloc(sizeof(hipfftComplex)*NX*BATCH); 
  if (redata == NULL)
    return 1;

  cxdata = (hipfftComplex*)malloc(sizeof(hipfftComplex)*NX*BATCH); 
  if (cxdata == NULL)
    return 1;

  for (i=0; i<NX*BATCH; i++){
    redata[i].x = sinf(2 * PI * i / NX);
  }

#if 0
  for (i=0; i<NX*BATCH; i++){
#ifdef DEBUG
    fprintf(stderr, "%0.2f ", redata[i]);
#endif
  }
#ifdef DEBUG
  fprintf(stderr, "\n");
#endif
#endif

  hipMalloc((void **) &idata, sizeof(hipfftComplex) * NX * BATCH);
  if (hipGetLastError() != hipSuccess){
#ifdef DEBUG
    fprintf(stderr, "cuda err: failed to allocate\n");
#endif
    free(redata);
    free(cxdata);
    return 1;
  }

  hipMalloc((void **) &odata, sizeof(hipfftComplex) * NX * BATCH);
  if (hipGetLastError() != hipSuccess){
#ifdef DEBUG
    fprintf(stderr, "cuda err: failed to allocate\n");
#endif
    hipFree(idata);
    free(redata);
    free(cxdata);
    return 1;
  }

  hipMemcpy(idata, redata, sizeof(hipfftComplex) * NX * BATCH, hipMemcpyHostToDevice);
  if (hipGetLastError() != hipSuccess){
#ifdef DEBUG
    fprintf(stderr, "cuda err: failed copy\n");
#endif
    hipFree(idata);
    hipFree(odata);
    free(redata);
    free(cxdata);
    return 1;
  }

  if (hipfftPlan1d(&plan, NX, HIPFFT_C2C, BATCH) != HIPFFT_SUCCESS) {
#ifdef DEBUG
    fprintf(stderr, "cuda err: plan creation failed\n");
#endif
    hipFree(idata);
    hipFree(odata);
    free(redata);
    free(cxdata);
    return 1;
  }
  
  hipfftResult res = hipfftExecC2C(plan, idata, odata, HIPFFT_FORWARD);
  if (res != HIPFFT_SUCCESS) {
#ifdef DEBUG
    fprintf(stderr ,"CUFFT error: ExecC2C Forward failed %d\n", res);
#endif
    hipfftDestroy(plan);
    hipFree(idata);
    hipFree(odata);
    free(redata);
    free(cxdata);
    return 1;
  }

  hipMemcpy(cxdata, odata, sizeof(hipfftComplex) * NX * BATCH, hipMemcpyDeviceToHost);

#if 0
  if (hipfftExecC2C(plan, data, data, HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
#ifdef DEBUG
    fprintf(stderr ,"CUFFT error: ExecC2C Forward failed");
#endif
    hipfftDestroy(plan);
    hipFree(idata);
    hipFree(odata);
    free(redata);
    free(cxdata);
    return 1;
  }
#endif
  
  if (hipDeviceSynchronize() != hipSuccess){
#ifdef DEBUG
    fprintf(stderr ,"CUFFT error: ExecC2C Forward failed");
#endif
    hipfftDestroy(plan);
    hipFree(idata);
    hipFree(odata);
    free(redata);
    free(cxdata);
    return 1;
  }

  
  for (i=0; i<NX*BATCH; i++){
#ifdef DEBUG
    fprintf(stderr, "%d %0.5f %0.5f %0.5f\n", i, redata[i].x, hipCabsf(cxdata[i]), atan2(cxdata[i].y, cxdata[i].x));
#endif
  }
#ifdef DEBUG
  fprintf(stderr, "\n");
#endif

  hipfftDestroy(plan);
  hipFree(idata);
  hipFree(odata);
  free(redata);
  free(cxdata);

  return 0;
}
#endif
