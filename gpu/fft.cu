#include "hip/hip_runtime.h"
/* (c) 2012 SKA SA */
/* Released under the GNU GPLv3 - see COPYING */

#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <math.h>
#include <fcntl.h>

#include <hipfft/hipfft.h>

#include <spead_api.h>

#define NX      64*1024
#define BATCH   1

#if 0
#define BUFFER  1
#endif

#define SPEAD_DATA_ID       0x0

struct cufft_o {
  hipfftHandle     plan;
  hipfftComplex    *d_host;
  hipfftComplex    *d_device;
  size_t          len;
#if 0
  float           *out;
  float           *h_out;
  struct spead_item_group *buf;
  int inum;
#endif
};

#if 0
static __global__ void power(hipfftComplex *data, float *out)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  out[i] = hipCabsf(data[i]);
}
static __global__ void phase(hipfftComplex *data, float *out)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  out[i] = atan2(data[i].y, data[i].x);
}
#endif

void spead_api_destroy(void *data)
{
  struct cufft_o *fo;

  fo = (struct cufft_o *)data;

  if (fo){
  
    hipfftDestroy(fo->plan);
    
    if (fo->d_host)
      free(fo->d_host);

#if 0
    if (fo->h_out)
      free(fo->h_out);
    hipFree(fo->out);
    destroy_item_group(fo->buf); 
#endif

    hipFree(fo->d_device);
  
    free(fo);
  }
}


void *spead_api_setup()
{
  struct cufft_o *fo;
  
  fo = (struct cufft_o*) malloc(sizeof(struct cufft_o));
  if (fo == NULL){
#ifdef DEBUG
    fprintf(stderr, "e: logic could not malloc api obj\n");
#endif
    return NULL;
  }
  
  fo->d_host  = NULL;
  fo->d_device = NULL;
  
  if (hipfftPlan1d(&(fo->plan), NX, HIPFFT_C2C, BATCH) != HIPFFT_SUCCESS) {
#ifdef DEBUG
    fprintf(stderr, "cuda err: plan creation failed\n");
#endif
    spead_api_destroy(fo);
    return NULL;
  }

  fo->len = sizeof(hipfftComplex)*NX*BATCH;

  fo->d_host = (hipfftComplex*) malloc(fo->len); 
  if (fo->d_host == NULL){
#ifdef DEBUG
    fprintf(stderr, "e: malloc failed\n");
#endif
    spead_api_destroy(fo);
    return NULL;
  }

#if 0
  fo->h_out = (float*) malloc(sizeof(float)*NX*BATCH); 
  if (fo->d_host == NULL){
#ifdef DEBUG
    fprintf(stderr, "e: malloc failed\n");
#endif
    spead_api_destroy(fo);
    return NULL;
  }

  hipMalloc((void **) &(fo->out), sizeof(float)*NX*BATCH); 
  if (hipGetLastError() != hipSuccess){
#ifdef DEBUG
    fprintf(stderr, "e: malloc failed\n");
#endif
    spead_api_destroy(fo);
    return NULL;
  }
#endif

  hipMalloc((void **) &(fo->d_device), fo->len);
  if (hipGetLastError() != hipSuccess){
#ifdef DEBUG
    fprintf(stderr, "cuda err: failed to cudamalloc\n");
#endif
    spead_api_destroy(fo);
    return NULL;
  }

#if 0
  fo->buf = create_item_group(BUFFER*NX*BATCH*sizeof(float2), BUFFER);
  if (fo->buf == NULL){
    spead_api_destroy(fo);
    return NULL;
  }

  fo->inum = 0;
#endif

  return fo;
}

int cufft_callback(struct cufft_o *fo, struct spead_api_item *itm)
{
  uint64_t i;

  uint8_t *d;  

  hipfftComplex *hst;
  hipfftComplex *dvc;

  hipfftResult res;
  
  if (fo == NULL || itm == NULL){
    return -1;
  }
 
  if (NX*BATCH != itm->i_data_len){
#ifdef DEBUG
    fprintf(stderr, "e: data len [%ld] doesn't match fft setup NX*BATCH [%ld]\n", itm->i_data_len, (long int) NX*BATCH);
#endif
    return -1;
  }

  hst  = fo->d_host;
  dvc = fo->d_device;
  d   = itm->i_data;

  if (hst == NULL || dvc == NULL || d == NULL){
#ifdef DEBUG
    fprintf(stderr, "e: data pointers are null\n");
#endif
    return -1;
  }

  /*prepare the data into hipComplex*/
  
  for (i=0; i<NX*BATCH; i++){
    hst[i].x = (float) d[i];
    hst[i].y = 0;
  }
  
  hipMemcpy(dvc, hst, fo->len, hipMemcpyHostToDevice);
  if (hipGetLastError() != hipSuccess){
#ifdef DEBUG
    fprintf(stderr, "cuda err: failed copy\n");
#endif
    return -1;
  }

  res = hipfftExecC2C(fo->plan, dvc, dvc, HIPFFT_FORWARD);
  if (res != HIPFFT_SUCCESS) {
#ifdef DEBUG
    fprintf(stderr ,"CUFFT error: ExecC2C Forward failed %d\n", res);
#endif
    return -1;  
  }


#if 0
  /*cuda kernel*/
  power<<<NX/1024, 1024>>>(dvc, fo->out);  

  hipMemcpy(fo->h_out, fo->out, sizeof(float)*NX*BATCH, hipMemcpyDeviceToHost);
#endif


  hipMemcpy(hst, dvc, fo->len, hipMemcpyDeviceToHost);
  if (hipGetLastError() != hipSuccess){
#ifdef DEBUG
    fprintf(stderr, "cuda err: failed copy\n");
#endif
    return -1;
  }


  if (set_spead_item_io_data(itm, hst, fo->len) < 0){
#ifdef DEBUG
    fprintf(stderr, "err: storeing cufft output\n");
#endif
    return -1;
  }


#if 0
  print_data( (unsigned char *) in, sizeof(hipfftComplex)*NX*BATCH);
  for (i=0; i<NX*BATCH; i++){
    fprintf(stderr, "%f + j %f\n", in[i].x, in[i].y);
  }
#endif

#if 0
  /*compute power*/
  for (i=0; i<NX*BATCH; i++){
#ifdef DEBUG
    fprintf(stdout, "%ld %0.5f\n", i, hipCabsf(in[i]));
#endif
  }
#ifdef DEBUG
  fprintf(stdout, "e\n");
#endif
  /*compute phase*/
  for (i=0; i<NX*BATCH; i++){
#ifdef DEBUG
    fprintf(stdout, "%ld %0.5f\n", i, atan2(in[i].y, in[i].x));
#endif
  }
#ifdef DEBUG
  fprintf(stdout, "e\n");
#endif
#endif

  return 0;
}


int spead_api_callback(struct spead_item_group *ig, void *data)
{
  struct spead_api_item *itm, *dst;
  struct cufft_o *fo;
  uint64_t off;
  int i, j;

  hipfftComplex *c;

  fo = (struct cufft_o *) data;


  if (fo == NULL || ig == NULL){
#ifdef DEBUG
    fprintf(stderr, "e: NULL params for <%s>\n", __func__);
#endif
    return -1;
  }
  

  off = 0;

  while (off < ig->g_size){

    itm = get_spead_item_at_off(ig, off);

#if 0
def DEBUG
    fprintf(stderr, "ITEM id[0x%x] vaild [%d] len [%ld]\n", itm->i_id, itm->i_valid, itm->i_len);
#endif

    if (itm == NULL)
      return -1;

    if (itm->i_id == SPEAD_DATA_ID){
      break;
    }

    off += sizeof(struct spead_api_item) + itm->i_len;
  }


  if (itm->i_id != SPEAD_DATA_ID){
#ifdef DEBUG
    fprintf(stderr, "%s: err dont have requested data id\n", __func__);
#endif
    return -1;
  }



  if (cufft_callback(fo, itm) < 0){
    return -1;
  }



#if 0
  dst = new_item_from_group(fo->buf, fo->len);
  if (dst == NULL){
#ifdef DEBUG
    fprintf(stderr, "%s: <%s> failed\n", __func__, __FILE__);
#endif
    return -1;
  }

  if (copy_to_spead_item(dst, fo->d_host, fo->len) < 0){
    return -1;
  }

  fo->inum++;
#endif

#if 0 
def DEBUG
  fprintf(stderr, "inum: %d\n", fo->inum);
#endif

#if 0
  itm = NULL;

  if (fo->buf == NULL)
    return -1;

  if (fo->inum == BUFFER){

    fprintf(stdout, "set term png size 1920,1080\nset view map\nsplot '-' matrix with image\n");
    
    off=0; 
    j  =0;
  
    while (off < fo->buf->g_size){
      itm = get_spead_item_at_off(ig, off);

#ifdef DEBUG
      fprintf(stderr, "ITEM [%d] id[0x%x] vaild [%d] len [%ld] of IGsize [%ld]\n", j++, itm->i_id, itm->i_valid, itm->i_len, fo->buf->g_size);
#endif
      if (itm->i_len > 0){
        c = (hipfftComplex *)(itm->i_data);
        for (i=NX*BATCH/2; i<NX*BATCH; i++){ 
          fprintf(stdout, "%0.5f ", hipCabsf(c[i]));
          //fprintf(stdout, "%0.5f ", atan2(c[i].y, c[i].x));
        }
        fprintf(stdout,"\n");
      }

      off += sizeof(struct spead_api_item) + itm->i_len;
    }
    fprintf(stdout,"e\ne\n");

  }
#endif

  return 0;
}


#if 0
#define PI (3.141592653589793)

__global__ void real2complex(float *f, hipfftComplex *fc, int N)
{
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  int j = threadIdx.y + blockIdx.y*blockDim.y;

  int index = j*N+i;

  if (i<N && j<N) {
    fc[index].x = f[index];
    fc[index].y = 0.0f;
  }
}

__global__ void complex2real(hipfftComplex *fc, float *f, int N)
{
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  int j = threadIdx.y + blockIdx.y*blockDim.y;

  int index = j*N+i;
  
  if (i<N && j<N) {
    f[index] = fc[index].x/((float)N*(float)N);
    //divide by number of elements to recover value
  }
}
#endif

#if 0
int main(int argc, char *argv[])
{
#define NX    256
#define BATCH 100

  int i; 

  hipfftHandle    plan;
  hipfftComplex  *odata;
  hipfftComplex  *cxdata;
  hipfftComplex  *redata;
  hipfftComplex  *idata;

  redata = (hipfftComplex*)malloc(sizeof(hipfftComplex)*NX*BATCH); 
  if (redata == NULL)
    return 1;

  cxdata = (hipfftComplex*)malloc(sizeof(hipfftComplex)*NX*BATCH); 
  if (cxdata == NULL)
    return 1;

  for (i=0; i<NX*BATCH; i++){
    redata[i].x = sinf(2 * PI * i / NX);
  }

#if 0
  for (i=0; i<NX*BATCH; i++){
#ifdef DEBUG
    fprintf(stderr, "%0.2f ", redata[i]);
#endif
  }
#ifdef DEBUG
  fprintf(stderr, "\n");
#endif
#endif

  hipMalloc((void **) &idata, sizeof(hipfftComplex) * NX * BATCH);
  if (hipGetLastError() != hipSuccess){
#ifdef DEBUG
    fprintf(stderr, "cuda err: failed to allocate\n");
#endif
    free(redata);
    free(cxdata);
    return 1;
  }

  hipMalloc((void **) &odata, sizeof(hipfftComplex) * NX * BATCH);
  if (hipGetLastError() != hipSuccess){
#ifdef DEBUG
    fprintf(stderr, "cuda err: failed to allocate\n");
#endif
    hipFree(idata);
    free(redata);
    free(cxdata);
    return 1;
  }

  hipMemcpy(idata, redata, sizeof(hipfftComplex) * NX * BATCH, hipMemcpyHostToDevice);
  if (hipGetLastError() != hipSuccess){
#ifdef DEBUG
    fprintf(stderr, "cuda err: failed copy\n");
#endif
    hipFree(idata);
    hipFree(odata);
    free(redata);
    free(cxdata);
    return 1;
  }

  if (hipfftPlan1d(&plan, NX, HIPFFT_C2C, BATCH) != HIPFFT_SUCCESS) {
#ifdef DEBUG
    fprintf(stderr, "cuda err: plan creation failed\n");
#endif
    hipFree(idata);
    hipFree(odata);
    free(redata);
    free(cxdata);
    return 1;
  }
  
  hipfftResult res = hipfftExecC2C(plan, idata, odata, HIPFFT_FORWARD);
  if (res != HIPFFT_SUCCESS) {
#ifdef DEBUG
    fprintf(stderr ,"CUFFT error: ExecC2C Forward failed %d\n", res);
#endif
    hipfftDestroy(plan);
    hipFree(idata);
    hipFree(odata);
    free(redata);
    free(cxdata);
    return 1;
  }

  hipMemcpy(cxdata, odata, sizeof(hipfftComplex) * NX * BATCH, hipMemcpyDeviceToHost);

#if 0
  if (hipfftExecC2C(plan, data, data, HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
#ifdef DEBUG
    fprintf(stderr ,"CUFFT error: ExecC2C Forward failed");
#endif
    hipfftDestroy(plan);
    hipFree(idata);
    hipFree(odata);
    free(redata);
    free(cxdata);
    return 1;
  }
#endif
  
  if (hipDeviceSynchronize() != hipSuccess){
#ifdef DEBUG
    fprintf(stderr ,"CUFFT error: ExecC2C Forward failed");
#endif
    hipfftDestroy(plan);
    hipFree(idata);
    hipFree(odata);
    free(redata);
    free(cxdata);
    return 1;
  }

  
  for (i=0; i<NX*BATCH; i++){
#ifdef DEBUG
    fprintf(stderr, "%d %0.5f %0.5f %0.5f\n", i, redata[i].x, hipCabsf(cxdata[i]), atan2(cxdata[i].y, cxdata[i].x));
#endif
  }
#ifdef DEBUG
  fprintf(stderr, "\n");
#endif

  hipfftDestroy(plan);
  hipFree(idata);
  hipFree(odata);
  free(redata);
  free(cxdata);

  return 0;
}
#endif
